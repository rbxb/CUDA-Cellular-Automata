#include "hip/hip_runtime.h"
#ifndef CONWAY_CU
#define CONWAY_CU

#include "hip/hip_runtime.h"
#include ""

#include "contants.h"
#include "helpers.cu"

namespace conway {

    // Kernel function for game of life transition
    __global__
    void transition(unsigned char* buf_r, unsigned char* buf_w) {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < SIZE) {
            int x = i % WIDTH;
            int y = i / WIDTH;
            unsigned char count = 0;
            count += buf_r[helpers::get_rel(x, y, 1, 1)] & 1;
            count += buf_r[helpers::get_rel(x, y, 1, 0)] & 1;
            count += buf_r[helpers::get_rel(x, y, 1, -1)] & 1;
            count += buf_r[helpers::get_rel(x, y, 0, 1)] & 1;
            count += buf_r[helpers::get_rel(x, y, 0, -1)] & 1;
            count += buf_r[helpers::get_rel(x, y, -1, 1)] & 1;
            count += buf_r[helpers::get_rel(x, y, -1, 0)] & 1;
            count += buf_r[helpers::get_rel(x, y, -1, -1)] & 1;
            if (count == 3) buf_w[i] = 255;
            else if (count != 2) buf_w[i] = 0;
            else buf_w[i] = buf_r[i];
        }
    }
};

#endif // CONWAY_CU