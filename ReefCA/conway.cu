#include "hip/hip_runtime.h"
/*
 * conway.cu
 *
 * https://github.com/rbxb/ReefCA
 */

#include "conway.cuh"

#include "cudahelpers.cuh"

using namespace ReefCA;

template<typename T>
__global__ void ReefCA::conway_transition(T* buf_r, T* buf_w, int width, int height, int depth) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < width * height) {
        int x = i % width;
        int y = i / height;
        unsigned char count = 0;
        count += buf_r[get_rel(x, y, 1, 1, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, 1, 0, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, 1, -1, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, 0, 1, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, 0, -1, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, -1, 1, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, -1, 0, width, height, depth)] & 1;
        count += buf_r[get_rel(x, y, -1, -1, width, height, depth)] & 1;
        if (count == 3) buf_w[i * depth] = 255;
        else if (count != 2) buf_w[i * depth] = 0;
        else buf_w[i * depth] = buf_r[i * depth];
    }
}

template<typename T>
__global__ void ReefCA::conway_transition_fast(T* buf_r, T* buf_w) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < WIDTHxHEIGHT) {
        int x = i % WIDTH;
        int y = i / HEIGHT;
        unsigned char count = 0;
        count += buf_r[get_rel(x, y, 1, 1)] & 1;
        count += buf_r[get_rel(x, y, 1, 0)] & 1;
        count += buf_r[get_rel(x, y, 1, -1)] & 1;
        count += buf_r[get_rel(x, y, 0, 1)] & 1;
        count += buf_r[get_rel(x, y, 0, -1)] & 1;
        count += buf_r[get_rel(x, y, -1, 1)] & 1;
        count += buf_r[get_rel(x, y, -1, 0)] & 1;
        count += buf_r[get_rel(x, y, -1, -1)] & 1;
        if (count == 3) buf_w[i * DEPTH] = 255;
        else if (count != 2) buf_w[i * DEPTH] = 0;
        else buf_w[i * DEPTH] = buf_r[i * DEPTH];
    }
}
