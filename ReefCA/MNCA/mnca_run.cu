#include <iostream>
#include <iomanip>
#include <vector>
#include <string>
#include <sstream>

#include "reefca.h"

#define FRAMES 200
#define SAVE_INTERVAL 1
#define THREADS 256

#define WIDTH 512
#define HEIGHT 512
#define DEPTH 1

const int SIZE = WIDTH * HEIGHT * DEPTH;

int main(void) {
    // Read MNCA rule to get neighborhood
    ReefCA::nhood* nhs;
    ReefCA::rule<unsigned char>* rules;
    int num_nhs;
    int num_rules;
    ReefCA::read_mnca_rule(&nhs, &num_nhs, &rules, &num_rules);

    // Allocate framebuffers
    unsigned char* buf_r;
    unsigned char* buf_w;
    hipMalloc(&buf_r, SIZE);
    hipMalloc(&buf_w, SIZE);

    // Allocate out buffer
    unsigned char* out_buffer = new unsigned char[SIZE];

    // Run seed noise kernel
    ReefCA::seed_wave<WIDTH, HEIGHT, DEPTH, unsigned char> << < (WIDTH * HEIGHT + THREADS - 1) / THREADS, THREADS >> > (buf_r, -1, 16);

    // Loop MNCA generations
    for (int i = 0; i < FRAMES; i++) {
        
        if (i % SAVE_INTERVAL == 0) {
            // Copy frame from device to host
            hipMemcpy(out_buffer, buf_r, SIZE, hipMemcpyDeviceToHost);

            // Wait for device to finish
            hipDeviceSynchronize();
        }

        // Start next transition
        ReefCA::mnca_transition<WIDTH, HEIGHT, DEPTH> 
            << < (WIDTH * HEIGHT + THREADS - 1) / THREADS, THREADS >> > 
            (buf_r, buf_w, nhs, rules, num_rules);

        // Update cout
        if (i % 10 == 0) {
            std::cout << i * 100 / FRAMES << "% \t" << i << " of " << FRAMES << std::endl;
        }

        if (i % SAVE_INTERVAL == 0) {
            // Save as PPM
            ReefCA::save_pam("out" + ReefCA::pad_image_index(i / SAVE_INTERVAL) + ".pam", out_buffer, WIDTH, HEIGHT, DEPTH);
        }
        
        // Swap buffers
        unsigned char* temp = buf_r;
        buf_r = buf_w;
        buf_w = temp;
    }


    // Save the final frame
    hipMemcpy(out_buffer, buf_r, SIZE, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    ReefCA::save_pam("out" + ReefCA::pad_image_index(FRAMES) + ".pam", out_buffer, WIDTH, HEIGHT, DEPTH);

    // Free GPU memory
    hipFree(buf_r);
    hipFree(buf_w);
    ReefCA::free_nhs_values(nhs, num_nhs);
    hipFree(nhs);
    hipFree(rules);

    std::cout << "Done!" << std::endl;

    return 0;
}