#include <iostream>
#include <iomanip>

#include "reefca.h"

#define FRAMES SIZE

int main(void) {
    unsigned char* buf_w;

    // Allocate buffers
    hipMalloc(&buf_w, SIZE);

    // Create out buffer
    unsigned char* out_buffer = new unsigned char[SIZE];

    // Create neighborhood
    std::vector<int> v = std::vector<int>();
    ReefCA::generate_nh_fill_circle(7, 3, v);
    nhood nh = ReefCA::upload_nh(v);

    // Draw neighborhood
    ReefCA::draw_nhood << < 1, 1 >> > (buf_w, 0, 0, nh);

    // Copy frame from device to host
    hipMemcpy(out_buffer, buf_w, SIZE, hipMemcpyDeviceToHost);

    // Wait for device to finish
    hipDeviceSynchronize();

    // Save as PPM
    ReefCA::save_pam("mnca_test.pam", out_buffer);
    
    // Free buffers
    hipFree(buf_w);
    hipFree(nh.p);

    return 0;
}