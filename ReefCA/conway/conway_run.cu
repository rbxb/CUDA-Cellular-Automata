#include <iostream>
#include <iomanip>

#include "conway.cu"
#include "helpers.cu"

#define FRAMES 100

int main(void) {
    char* buf_r;
    char* buf_w;

    // Allocate buffers
    hipMalloc(&buf_r, SIZE);
    hipMalloc(&buf_w, SIZE);

    // Create out buffer
    char* out_buffer = new char[SIZE];

    // Run seed kernel
    seed << < (SIZE + THREADS - 1) / THREADS, THREADS >> > (buf_r);

    // Loop conways game of life
    for (int i = 0; i < FRAMES; i++) {
        // Copy frame from device to host
        hipMemcpy(out_buffer, buf_r, SIZE, hipMemcpyDeviceToHost);

        // Wait for device to finish
        hipDeviceSynchronize();

        // Start next transition
        transition << < (SIZE + THREADS - 1) / THREADS, THREADS >> > (buf_r, buf_w);

        // Update cout
        if (i % 10 == 0) {
            std::cout << i * 100 / FRAMES << "% \t" << i << " of " << FRAMES << std::endl;
        }

        // Save as PPM
        save_image("out-" + std::to_string(i) + ".pam", out_buffer, WIDTH, HEIGHT, 1);

        // Swap buffers
        char* temp = buf_r;
        buf_r = buf_w;
        buf_w = temp;
    }


    // Save the final frame
    hipMemcpy(out_buffer, buf_r, SIZE, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    save_image("out-" + std::to_string(FRAMES) + ".pam", out_buffer, WIDTH, HEIGHT, 1);

    // Free buffers
    hipFree(buf_r);
    hipFree(buf_w);

    std::cout << "Done!" << std::endl;

    return 0;
}